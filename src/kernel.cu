#include "hip/hip_runtime.h"
// -*- mode: c -*-

#include <stdio.h>
#include <float.h>

#include "point.h"
#include "cuda_util.h"
#include "constants.h"

// block sizes
#define DISTANCE_BLOCK_SIZE 32
#define SUM_BLOCK_SIZE 256 // can't make this too big, because of limited __shared__ memory

//Clustering Calculation
__global__
void distances_k(Point* points_d, int points_size, Point* centroids_d) {
    //Get the index for the current point to work with
    int index = threadIdx.x + blockIdx.x * blockDim.x;

    if(index < points_size) {
        Point* p = &points_d[index];
        for (int cluster_id = 0; cluster_id < K_CLUSTERS; cluster_id++) {
            Point c = centroids_d[cluster_id];
            // can't call non-kernel function from kernel function, so we do this here.
            double dist = ((c.x - p->x) * (c.x - p->x) + (c.y - p->y) * (c.y - p->y) + (c.z - p->z) * (c.z - p->z));// point_distance(c, *p);
            if (dist < p->min_dist) {
                p->min_dist = dist;
                p->cluster = cluster_id;
            }
        }
    }
}

// Do sum and reduce of the points x, y, and z values in one block
__global__
void sum_reduce_kernel(Point* points_d, int points_size, int* n_points_d, double* sum_x_d, double* sum_y_d, double* sum_z_d) {
    /// initialize variables
    int idx = threadIdx.x;

    __shared__ int n_points[SUM_BLOCK_SIZE][K_CLUSTERS];
    __shared__ double sum_x[SUM_BLOCK_SIZE][K_CLUSTERS];
    __shared__ double sum_y[SUM_BLOCK_SIZE][K_CLUSTERS];
    __shared__ double sum_z[SUM_BLOCK_SIZE][K_CLUSTERS];

    for (int cluster_id = 0; cluster_id < K_CLUSTERS; cluster_id++) {
        n_points[idx][cluster_id] = 0;
        sum_x[idx][cluster_id] = 0;
        sum_y[idx][cluster_id] = 0;
        sum_z[idx][cluster_id] = 0;
    }

    // compute the local sum
    for (int i = idx; i < points_size; i += blockDim.x) {
        Point* p = &points_d[i];
        n_points[idx][p->cluster] += 1;
        sum_x[idx][p->cluster] += p->x;
        sum_y[idx][p->cluster] += p->y;
        sum_z[idx][p->cluster] += p->z;

        p->min_dist = DBL_MAX;
    }

    __syncthreads();

    // do the reduction
    for (int size = blockDim.x / 2; size > 0; size /= 2) {
        if (idx < size) {
            for (int cluster_id = 0; cluster_id < K_CLUSTERS; cluster_id++) {
                n_points[idx][cluster_id] += n_points[idx + size][cluster_id];
                sum_x[idx][cluster_id] += sum_x[idx + size][cluster_id];
                sum_y[idx][cluster_id] += sum_y[idx + size][cluster_id];
                sum_z[idx][cluster_id] += sum_z[idx + size][cluster_id];
            }
        }
        __syncthreads();
    }

    // put into output buffer
    if (idx == 0) {
        for (int cluster_id = 0; cluster_id < K_CLUSTERS; ++cluster_id) {
            n_points_d[cluster_id] = n_points[0][cluster_id];
            sum_x_d[cluster_id] = sum_x[0][cluster_id];
            sum_y_d[cluster_id] = sum_y[0][cluster_id];
            sum_z_d[cluster_id] = sum_z[0][cluster_id];
        }
    }
}

extern "C" void cuda_setup(Point* points_h, Point** points_d, int points_size, Point** centroids_d, int** n_points_d, double** sum_x_d, double** sum_y_d, double** sum_z_d) {
    //Allocate device pointers and copy them to the device
    checkCuda(hipMalloc((void **) points_d, sizeof(Point)*points_size));
    checkCuda(hipMalloc((void **) centroids_d, sizeof(Point)*K_CLUSTERS)); // just allocate the memory now, we will memcpy every kernel call
    checkCuda(hipMalloc((void **) n_points_d, sizeof(int)*K_CLUSTERS));
    checkCuda(hipMalloc((void **) sum_x_d, sizeof(double)*K_CLUSTERS));
    checkCuda(hipMalloc((void **) sum_y_d, sizeof(double)*K_CLUSTERS));
    checkCuda(hipMalloc((void **) sum_z_d, sizeof(double)*K_CLUSTERS));
    checkCuda(hipDeviceSynchronize());

    checkCuda(hipMemcpy(*points_d, points_h, sizeof(Point)*points_size, hipMemcpyHostToDevice));
    checkCuda(hipDeviceSynchronize());
}

extern "C" void cuda_cleanup(Point* points_h, Point* points_d, int points_size, Point* centroids_d, int* n_points_d, double* sum_x_d, double* sum_y_d, double* sum_z_d) {
    //copy device points to host points
    checkCuda(hipMemcpy(points_h, points_d, sizeof(Point)*points_size, hipMemcpyDeviceToHost));
    checkCuda(hipDeviceSynchronize());

    //Free device pointers
    checkCuda(hipFree(points_d));
    checkCuda(hipFree(centroids_d));
    checkCuda(hipFree(n_points_d));
    checkCuda(hipFree(sum_x_d));
    checkCuda(hipFree(sum_y_d));
    checkCuda(hipFree(sum_z_d));
    checkCuda(hipDeviceSynchronize());
}

// Function that launches the CUDA kernel
extern "C" void cuda_distances_kernel(Point* points_d, int points_size, Point* centroids_h, Point* centroids_d) {
    dim3 DimGrid(ceil(points_size/((float)DISTANCE_BLOCK_SIZE)));
    dim3 DimBlock(DISTANCE_BLOCK_SIZE);

    // copy the centroids over
    checkCuda(hipMemcpy(centroids_d, centroids_h, sizeof(Point)*K_CLUSTERS, hipMemcpyHostToDevice));
    checkCuda(hipDeviceSynchronize());

    //Launch kernel
    distances_k<<<DimGrid, DimBlock>>>(points_d, points_size, centroids_d);
    checkCuda(hipDeviceSynchronize());
}

extern "C" void cuda_sum_kernel(Point* points_d,
                                int points_size,
                                int* n_points_h,
                                double* sum_x_h,
                                double* sum_y_h,
                                double* sum_z_h,
                                int* n_points_d,
                                double* sum_x_d,
                                double* sum_y_d,
                                double* sum_z_d) {
    // do the computation
    sum_reduce_kernel<<<1, SUM_BLOCK_SIZE>>>(points_d, points_size, n_points_d, sum_x_d, sum_y_d, sum_z_d);
    checkCuda(hipDeviceSynchronize());

    checkCuda(hipMemcpy(n_points_h, n_points_d, sizeof(int)*K_CLUSTERS, hipMemcpyDeviceToHost));
    checkCuda(hipMemcpy(sum_x_h, sum_x_d, sizeof(double)*K_CLUSTERS, hipMemcpyDeviceToHost));
    checkCuda(hipMemcpy(sum_y_h, sum_y_d, sizeof(double)*K_CLUSTERS, hipMemcpyDeviceToHost));
    checkCuda(hipMemcpy(sum_z_h, sum_z_d, sizeof(double)*K_CLUSTERS, hipMemcpyDeviceToHost));
    checkCuda(hipDeviceSynchronize());
}
